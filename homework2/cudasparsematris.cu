#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "../matrix/definitions.h"

__global__ void multi(int *D_array, int *Rp_array, int *C_array, int *sonuc)//Grafik kart üzerinde koşacak kod
{
	int cvalue = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	Rp_array[counter+1],D_array[j]sonuc[0][k-1]V[0][counter]
	if (row > DIM || col > DIM) return;

	for (int k = 0; k < col; k++) {
	  int satir = 0,counter=0;
	 for (int  j = 0; j < row; j++) {
	   if (C_array[j]==k) {
	    while(satir < j){
	      satir = satir + (Rp_array[counter+1]-Rp_array[counter]);
	      counter++;
	    }
	    sonuc[0][k-1] += D_array[j] * V[0][counter];
	    }
	   }
	  }

	C[row*DIM + col] = cvalue;
}

void matrixmulti(int *D_array, int *Rp_array, int *C_array, int *sonuc){
	int *dev_a, *dev_b, *dev_c, *sonuc;

	hipEvent_t start, stop;
	hipEventCreate(&start);

	//allocate memory on global memory of gpu

	FILE *fp_C,*fp_D,*fp_Rp;
		 fp_C = fopen("C.txt", "r");
		 double C_array[99911];
		 for (int i = 0; i < 99911 ; i++) {
				 fscanf(fp_C, "%lf", &C_array[i]);
		 }
		 fclose(fp_C);

	hipError_t err = hipMalloc((void**)&fp_C, ((DIM)*(DIM))*sizeof(int));
	err = hipMemcpy(fp_C, C_array, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);

	fp_D = fopen("D.txt","r");
	double D_array[99911];
	for (int j = 0; j < 99911; j++) {
		fscanf(fp_D,"%lf",&D_array[j]);
	}

	fclose(fp_D);
	hipError_t err = hipMalloc((void**)&fp_D, ((DIM)*(DIM))*sizeof(int));
	err = hipMemcpy(fp_D, D_array, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);


	int Rp_array[10001];
	 fp_Rp = fopen("Rp.txt", "r");
	for (int i = 0; i < 10001 ; i++) {
			fscanf(fp_Rp, "%d", &Rp_array[i]);
	}

	hipError_t err = hipMalloc((void**)&fp_Rp, ((DIM)*(DIM))*sizeof(int));
	err = hipMemcpy(fp_Rp, Rp_array, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);

	fclose(fp_Rp);
	//veri kopyala gpu'da  koş veriyi geri al
	// Saat bulma
	//two dimension threads
	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);

	hipEventRecord(start);
	multi << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);
	hipEventRecord(stop);

	err = hipMemcpy(sonuc, sonuc, ((DIM*DIM))*sizeof(int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));

	//call the kernel function multi
	hipEventRecord(start);
	multi << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);
	hipEventRecord(stop);

	for (int i = 0; i < 10000; i++) {
	  /* code */
	  fprintf(f,"%.3f ",sonuc[0][i]);
	}
	 fclose(f);

	//free the memory

	hipFree(D_array);
	hipFree(Rp_array);
	hipFree(C_array);
	hipFree(sonuc);

	hipEventCreate(&stop);
}
